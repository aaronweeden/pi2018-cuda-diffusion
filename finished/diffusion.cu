#include "hip/hip_runtime.h"
/*******************************************************************************
 * Diffusion Model
 *
 * A 2D rectangular world is filled with cells. Each of the four edges of the
 * world (top, left, right, and bottom) has its own unchanging value. The
 * middle of the world starts with an initial value, but as the model advances,
 * the value of each cell in the middle changes by setting itself equal to
 * the average of its four neighbors to the top, left, right, and bottom.
 * The model advances a certain number of time steps.
 *
 * The user can provide command line options in order to set the model
 * parameters. These are listed in the ParseArgs function.
 *
 * (c) Shodor, 2017
 ******************************************************************************/

/*************
 * Libraries *
 *************/
#include <omp.h> // For omp_get_wtime()
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

/********************
 * Global variables *
 ********************/
int IsPrinting = true;
int NumRows = 10;
int NumCols = 10;
int NumSteps = 100;
float TopVal = 100.0;
float BottomVal = 0.0;
float LeftVal = 0.0;
float RightVal = 0.0;
float MiddleVal = 0.0;
int OutputPrecision = 3;
int NumCellsWithBounds;    // Calculated later using CalcNumCells()
int NumCellsWithoutBounds; // Calculated later using CalcNumCells()
int CellCharSize;          // Calculated later using CalcCellCharSize()
float * CellsWithBounds;   // Array of cell values, including the t,l,r,b bounds
float * CellsWithoutBounds; // Array of cell values, not including the bounds.
// We need 2 arrays because we do not want to "check" and "change" in the same
// loop; if a cell is updated before its neighbor checks its value, we will
// get the wrong result. See the CalcCells() and CopyCells() functions.
// We only need one of the arrays to have the cell boundaries; we can save
// memory with the other one by not including them.
float * d_CellsWithBounds;
float * d_CellsWithoutBounds;

/**********************
 Function definitions *
 **********************/
// Check whether a CUDA call was successful
void TryCuda(hipError_t const err)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Check the command line arguments to see if the user provided any model
// parameters
void ParseArgs(int argc, char ** argv)
{
  char c;

  // See the documentation for getopt()
  while ((c = getopt(argc, argv, "qw:h:s:t:l:r:b:p:")) != -1)
  {
    switch(c)
    {
      case 'q':
        IsPrinting = false;
        break;
      case 'w':
        NumCols = atoi(optarg);
        break;
      case 'h':
        NumRows = atoi(optarg);
        break;
      case 's':
        NumSteps = atoi(optarg);
        break;
      case 't':
        TopVal = atof(optarg);
        break;
      case 'l':
        LeftVal = atof(optarg);
        break;
      case 'r':
        RightVal = atof(optarg);
        break;
      case 'b':
        BottomVal = atof(optarg);
        break;
      case 'p':
        OutputPrecision = atoi(optarg);
        break;
      case '?':
      default:
        fprintf(stderr, "Usage: %s [OPTIONS]\n"
          "OPTIONS:\n"
          "-q : turn off output\n"
          "-h <arg> : set number of rows\n"
          "-w <arg> : set number of columns\n"
          "-s <arg> : set number of time steps\n"
          "-t <arg> : set value of top boundary\n"
          "-l <arg> : set value of left boundary\n"
          "-r <arg> : set value of right boundary\n"
          "-b <arg> : set value of bottom boundary\n"
          "-p <arg> : set output precision\n",
          argv[0]);
    }
  }
}

// Calculate the number of cells in the world
void CalcNumCells()
{
  NumCellsWithBounds = (NumRows + 2) * (NumCols + 2);
  NumCellsWithoutBounds = NumRows * NumCols;
}

// Calculate the number of characters to use in displaying a single heat value
void CalcCellCharSize()
{
  int numDigits;

  // Initialize
  CellCharSize = 0;

  // If the top bound has the biggest value so far, use it
  if (TopVal > CellCharSize)
  {
    CellCharSize = TopVal;
  }

  // If the left bound has the biggest value so far, use it
  if (LeftVal > CellCharSize)
  {
    CellCharSize = LeftVal;
  }

  // If the right bound has the biggest value so far, use it
  if (RightVal > CellCharSize)
  {
    CellCharSize = RightVal;
  }

  // If the bottom bound has the biggest value so far, use it
  if (BottomVal > CellCharSize)
  {
    CellCharSize = BottomVal;
  }

  // Count the number of digits
  numDigits = 0;
  while (CellCharSize != 0)
  {
    CellCharSize /= 10;
    numDigits++;
  }
  CellCharSize = numDigits;

  // Add extra characters for the decimal point and the digits after it
  CellCharSize += 1 + OutputPrecision;
}

// Check if a call to malloc() was successful by examining its return value
void CheckMalloc(void * const val)
{
  if (val == NULL)
  {
    fprintf(stderr, "malloc error\n");
    exit(EXIT_FAILURE);
  }
}

// Allocate memory for dynamic arrays of cell values
void AllocMemory()
{
  // Allocate memory on the host
  CellsWithBounds    = (float *)malloc(NumCellsWithBounds * sizeof(float));
  CheckMalloc(CellsWithBounds);

  CellsWithoutBounds = (float *)malloc(NumCellsWithoutBounds *
    sizeof(float));
  CheckMalloc(CellsWithoutBounds);

  // Allocate memory on the device
  TryCuda(hipMalloc((void**)&d_CellsWithBounds, NumCellsWithBounds *
    sizeof(float)));
  TryCuda(hipMalloc((void**)&d_CellsWithoutBounds, NumCellsWithoutBounds *
    sizeof(float)));
}

// Set the initial cell values
void InitializeArrays()
{
  int row;
  int col;

  for (col = 0; col < NumCols; col++)
  {
    // Set the top bound
    CellsWithBounds[col + 1] = TopVal;

    // Set the bottom bound
    CellsWithBounds[NumRows * (NumCols + 2) + col + 1] = BottomVal;
  }

  for (row = 0; row < NumRows; row++)
  {
    // Set the left bound
    CellsWithBounds[(row + 1) * (NumCols + 2) + 0] = LeftVal;

    // Set the middle
    for (col = 0; col < NumCols; col++)
    {
      CellsWithBounds[(row + 1) * (NumCols + 2) + col + 1] = MiddleVal;
    }

    // Set the right bound
    CellsWithBounds[(row + 1) * (NumCols + 2) + NumCols] = LeftVal;
  }
}

// Print the values of each cell to the standard output
void PrintCells(int const time)
{
  int row;
  int col;

  printf("Time step %d\n", time);

  for (row = 0; row < NumRows; row++)
  {
    for (col = 0; col < NumCols; col++)
    {
      printf("%*.*f ", CellCharSize, OutputPrecision,
        CellsWithoutBounds[row * NumCols + col]);
    }
    printf("\n");
  }
  printf("\n");
}

// Calculate the average of nearest-neighbors for each cell. Make sure not
// to check and change values in the same array, or the calculation will be
// wrong.
void CalcCells()
{
  CalcCells_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (d_CellsWithBounds, d_CellsWithoutBounds, NumCellsWithBounds, NumCols);
}

__global__ void CalcCells_kernel(float * const d_CellsWithBounds,
  float * const d_CellsWithoutBounds, int const NumCellsWithBounds,
  int const NumCols)
{
  // Calculate the unique ID, row, and column for the current CUDA thread
  int const threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int const row = threadId / NumCols;
  int const col = threadId % NumCols;

  // All threads whose thread ID is >= the count will NOT do the following,
  // thus avoiding writing into un-allocated space.
  if (threadId < NumCellsWithBounds)
  {
    // Start with nothing
    d_CellsWithoutBounds[row * NumCols + col] = 0.0;

    // Add the top neighbor
    d_CellsWithoutBounds[row * NumCols + col] +=
      d_CellsWithBounds[row * (NumCols + 2) + col + 1];

    // Add the left neighbor
    d_CellsWithoutBounds[row * NumCols + col] +=
      d_CellsWithBounds[(row + 1) * (NumCols + 2) + col];

    // Add the right neighbor
    d_CellsWithoutBounds[row * NumCols + col] +=
      d_CellsWithBounds[(row + 1) * (NumCols + 2) + col + 2];

    // Add the bottom neighbor
    d_CellsWithoutBounds[row * NumCols + col] +=
      d_CellsWithBounds[(row + 2) * (NumCols + 2) + col + 1];

    // Divide by 4 (multiplication is a bit faster, so multiply by 1/4)
    d_CellsWithoutBounds[row * NumCols + col] *= 0.25;
  }
}

// Make sure both arrays have the new average of nearest-neighbors for each cell
void CopyCells()
{
  CopyCells_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (d_CellsWithBounds, d_CellsWithoutBounds, NumCellsWithBounds, NumCols);
}

__global__ void CopyCells_kernel(float * const d_CellsWithBounds,
  float * const d_CellsWithoutBounds, int const NumCellsWithBounds,
  int const NumCols)
{
  // Calculate the unique ID, row, and column for the current CUDA thread
  int const threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int const row = threadId / NumCols;
  int const col = threadId % NumCols;

  d_CellsWithBounds[(row + 1) * (NumCols + 2) + col + 1] =
    d_CellsWithoutBounds[row * NumCols + col];
}

// Run the simulation
void Simulate()
{
  int time;

  // Copy data from the host to the device
  TryCuda(hipMemcpy(d_CellsWithBounds, CellsWithBounds,
    NumCellsWithBounds * sizeof(float), hipMemcpyHostToDevice));
  TryCuda(hipMemcpy(d_CellsWithoutBounds, CellsWithoutBounds,
    NumCellsWithoutBounds * sizeof(float), hipMemcpyHostToDevice));

  for (time = 0; time < NumSteps; time++)
  {
    if (IsPrinting)
    {
      // Copy data from the device to the host so it can printed
      TryCuda(hipMemcpy(CellsWithoutBounds, d_CellsWithoutBounds,
        NumCellsWithoutBounds * sizeof(float), hipMemcpyDeviceToHost));

      // Print the values of each cell to the standard output
      PrintCells(time);
    }

    // Calculate the average of nearest-neighbors for each cell. Make sure not
    // to check and change values in the same array, or the calculation will be
    // wrong.
    CalcCells();

    // Make sure both arrays have the new average of nearest-neighbors for each
    // cell
    CopyCells();
  }
}

// De-allocate memory for dynamic arrays of cell values
void FreeMemory()
{
  // Free the device memory
  TryCuda(hipFree(d_CellsWithoutBounds));
  TryCuda(hipFree(d_CellsWithBounds));

  // Free the host memory
  free(CellsWithoutBounds);
  free(CellsWithBounds);
}

// The main function, where program execution starts
int main(int argc, char ** argv)
{
  // Start a timer
  double startTime = omp_get_wtime();

  // Check the command line arguments to see if the user provided any model
  // parameters
  ParseArgs(argc, argv);

  // Calculate the number of cells in the world
  CalcNumCells();

  // Calculate the number of characters to use in displaying a single heat value
  CalcCellCharSize();

  // Allocate memory for dynamic arrays of cell values
  AllocMemory();

  // Set the initial cell values
  InitializeArrays();

  // Run the simulation
  Simulate();

  // De-allocate memory for dynamic arrays of cell values
  FreeMemory();

  // Stop the timer
  printf("Runtime: %f seconds\n", omp_get_wtime() - startTime);
}

